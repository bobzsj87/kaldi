#include "hip/hip_runtime.h"
// decoder/cuda-lattice-decoder.cu

// Copyright      2018  Zhehuai Chen

// See ../../COPYING for clarification regarding multiple authors
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
// http:// www.apache.org/licenses/LICENSE-2.0
//
// THIS CODE IS PROVIDED *AS IS* BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, EITHER EXPRESS OR IMPLIED, INCLUDING WITHOUT LIMITATION ANY IMPLIED
// WARRANTIES OR CONDITIONS OF TITLE, FITNESS FOR A PARTICULAR PURPOSE,
// MERCHANTABLITY OR NON-INFRINGEMENT.
// See the Apache 2 License for the specific language governing permissions and
// limitations under the License.

#include "lat/determinize-lattice-pruned.h"
#include "lat/kaldi-lattice.h"
#include "itf/decodable-itf.h"

#include "cuda-decoder-utils.h"
// #include "lattice-faster-decoder-cuda.h"
#include "decoder/cuda-lattice-decoder.h"


namespace kaldi {

typedef CudaLatticeDecoder::Token Token;
typedef CudaLatticeDecoder::StateId StateId;
typedef CudaLatticeDecoder::TokenState TokenState;
typedef CudaLatticeDecoder::CostType CostType;
typedef CudaLatticeDecoder::TokenLookupElem TokenLookupElem;
typedef CudaLatticeDecoder::LatLink LatLink;
typedef CudaLatticeDecoder::LatLinkCompact LatLinkCompact;
typedef CudaLatticeDecoder::LatLinkVector LatLinkVector;
typedef CudaLatticeDecoder::TokenMergeVector TokenMergeVector;
typedef CudaLatticeDecoder::processTokens_params processTokens_params;
typedef CudaLatticeDecoder::LatticeProcessor LatticeProcessor;
#define CudaVector CudaLatticeDecoder::CudaVector
#define CudaMergeVector CudaLatticeDecoder::CudaMergeVector

// instantiation of templates
template HOST DEVICE LatLink& CudaVector<LatLink>::operator[](uint32 idx);
template HOST DEVICE TokenState& CudaVector<TokenState>::operator[](uint32 idx);
template HOST DEVICE uint32  CudaVector<TokenState>::Size() const;
template HOST DEVICE uint32  CudaVector<LatLink>::Size() const;
template<> DEVICE void CudaMergeVector<TokenState>::StoreDataByPackIdx(
  void* temp_data_buf, int* temp_data_buf_update, int32 buf_size,
  LatticeProcessor *lattice_processor);

// inline functions

// swap code in device, as we need to instantiate it, so we define it here
template<typename T>
inline DEVICE void cuda_swap(T &a, T &b) {
  T c = a;
  a = b;
  b = c;
}

// device functions called by __global__ functions 

// during token passing, re-initialize lookuptables of visited states 
// to prepare for the next lookup
DEVICE static inline void _initialize_visited_states(
                                      TokenLookupElem *current_tokens_lookup,
                                      TokenMergeVector cur_toks) {
  int32 size = cur_toks.Size();
  for (int32 i = blockIdx.x * blockDim.x + threadIdx.x; i < size;
       i += blockDim.x * gridDim.x) {
    StateId state = cur_toks[i].state; // cur_toks will be clear in PreProcessTokens
    current_tokens_lookup[state].tokenstate_idx = LOOKUP_DEACTIVE;
    // do not need to clear Token* cur_tok, as data will be re-stored in it
    // in StoreDataByPackIdx()
  }
}

// initialize cutoff used in beam pruning
DEVICE static inline void _initialize_cutoff(CostType *cutoff) { *cutoff = 1e2; }

// _find_or_add_token_arc either locates a token in TokenLookupElem
// or if necessary add a token by activating it in TokenLookupElem
// for the current frame.  The function also adds a lattice arc into the vector
// it's a GPU version of FindOrAddToken() and ForwardLink()
DEVICE static inline void _find_or_add_token_arc(processTokens_params* params,
    StateId nextstate, CostType total_cost, CostType acoustic_cost,
    TokenState* ts, uint32 j, bool add_arc, TokenState** next_ts,
    uint64 **token_pack, int32* update_idx, bool is_emit) {
  TokenLookupElem& lookup_elem = params->current_tokens_lookup[nextstate];
  // check if token is active or not.  if not, activate it
  if (lookup_elem.tokenstate_idx == LOOKUP_DEACTIVE
      && atomicCAS((int32 *)&lookup_elem.tokenstate_idx,
                   (int32)LOOKUP_DEACTIVE, (int32)LOOKUP_READY_PUSH) ==
      (int32)LOOKUP_DEACTIVE) {
    // grab sentinal to see who gets to add to cur_toks list
    // if haven't seen this token, add into hash by activating it
    // push back the TokenState, and also record its index in lookup table
    int32 tokenstate_idx = params->cur_toks.PushBack(TokenState(nextstate));
    // firstly store it, so that other threads can continue processing
    lookup_elem.tokenstate_idx = tokenstate_idx;
    int32 tok_idx_allocated =
      params->lattice_processor.GetTokenAllocIdx(tokenstate_idx);
    // do not need to clear Token* cur_tok, as data will be stored in it
    // in function StoreDataByPackIdx()
    // use tokenstate_idx + front_d (accumulate number of tokens before
    // this frame) as tok_idx_allocated
    params->cur_toks[tokenstate_idx].tok_idx_allocated = tok_idx_allocated;
    // thus we do not need to ensure tok_idx_allocated has been allocated,
    // as it will be used after this iteration and at StoreDataByPackIdx()
  }
  // need both 2 steps below, to ensure tokenstate_idx recorded correctly
  while (lookup_elem.tokenstate_idx == LOOKUP_DEACTIVE ||  // hasn't pushed
         lookup_elem.tokenstate_idx == LOOKUP_READY_PUSH);  // during pushing
  __threadfence();

  *next_ts = &params->cur_toks[lookup_elem.tokenstate_idx]; // get it using index
  if (add_arc) { // we add lattice arc except in _add_initial_token()
    Token *prev_tok = params->lattice_processor.GetTokenByExactIdx(
                        ts->tok_idx_allocated);
    int32 prev_tok_frame = (!is_emit) ? params->frame : params->frame - 1;
    int32 ts_id = (!is_emit) ?
                  params->cur_toks.GetIdxFromAddr(ts) : // process non-emit tokens
                  params->prev_toks.GetIdxFromAddr(ts); // process emit tokens
    LatLinkCompact arc(ts_id, prev_tok_frame,
                       lookup_elem.tokenstate_idx, params->frame,
                       acoustic_cost, j);
    // use pushBack (idx - start index of this frame)
    // as update item index because it is unique in each frame obtained from
    // atomicAdd in pushBack function (lat_arcs_sub_vec
    // is recorded accumulatively and cleared only at end of decoding)
    int32 ret = params->lat_arcs_sub_vec.PushBack(arc) -
                  *params->num_arcs_till_last;
    assert(ret < params->max_lat_arc_per_frame);
    *update_idx = ret;
  }
  // get token_pack variable address for atomic based token recombination
  *token_pack = &((*next_ts)->token_pack);
  return;
}

DEVICE static inline void _find_prev_cutoff_by_histogram(
                          processTokens_params* params) {
  // to reduce a grid sync, we initialize it in _process_tokens()
  // params->histogram_prev_toks.Initialize(*params->cutoff - params->beam);
  bool rank0 = blockIdx.x == 0 && threadIdx.x == 0;
  int32 hist_local[MAX_HISTOGRAM_SIZE];
  memset(hist_local, 0, params->histogram_prev_toks.Size());

  int32 size = params->prev_toks.Size();
  for (int32 i = threadIdx.x + blockIdx.x * blockDim.x;
       i < size; i += gridDim.x * blockDim.x) {
    TokenState ts = params->prev_toks[i];
    Token * tok = params->lattice_processor.GetTokenByExactIdx(ts.tok_idx_allocated);
    params->histogram_prev_toks.AddScore2LocalHist(tok->cost_, hist_local);
  }
  params->histogram_prev_toks.AggregateLocalHist(hist_local);

  grid_sync(params->barrier);

  if (rank0)
    params->histogram_prev_toks.GetCutoff(params->cutoff_prev,
                                          params->max_active, params->verbose);
}

template<int32 blockDimx, int32 blockDimy>
DEVICE static inline void _find_best_cutoff(processTokens_params* params) {
  // blockDim threads per token to process out-arcs in parallel
  auto group = cooperative_groups::tiled_partition<blockDimx>
               (cooperative_groups::this_thread_block());
  CostType local_cutoff = INFINITY;
  int32 size = params->prev_toks.Size();

  // frame 0 don't obtain params->cutoff
  if (size > params->max_active && params->frame > 1) {
    _find_prev_cutoff_by_histogram(params);
    // params->cutoff_prev to be used in the latter part
    grid_sync(params->barrier);
  }


  // uses dynamically load balanced loop trips.  Tokens are assigned
  // dynamically instead of statically. details are described in
  // _process_emitting_tokens()
  while (true) {
    int32 i;
    if (group.thread_rank() == 0) { // thread 0 nominated to get new token
      i = atomicAdd(params->fb_idx, 1); // allocate new token index
    }
    i = group.shfl(i, 0); // rank 0 broadcasts i to whole group
    if (i >= size) break; // all tokens processed

    TokenState ts = params->prev_toks[i];
    Token * tok = params->lattice_processor.GetTokenByExactIdx(ts.tok_idx_allocated);

    if (size > params->max_active && tok->cost_ > *params->cutoff_prev)
      continue; // histogram pruning for last frame

    StateId state = ts.state;
    uint32 start = params->e_offsets[state], finish = params->e_offsets[state + 1];
    int32 ilabel, ilabel_next;
    int32 j = start + group.thread_rank();
    if (j < finish) {
      ilabel_next = params->arc_ilabels[j];
    }
    int32 nextj;

    for (j; j < finish; j = nextj) {  // thread parallelism
      nextj = j + blockDimx;
      ilabel = ilabel_next;
      if (nextj < finish) {
        ilabel_next = params->arc_ilabels[nextj];
      }

      BaseFloat acoustic_cost = -params->loglikelihoods[ilabel];
      CostType weight = params->arc_weights[j];
      CostType total_cost = tok->cost_ + weight + acoustic_cost + params->beam;

      if (total_cost < local_cutoff)
        local_cutoff = total_cost;
    }
  }

  // TODO: reduce inside block first
  if (local_cutoff != INFINITY) {
    atomic_min(params->cutoff, local_cutoff);
  }
}

template<int32 blockDimx, int32 blockDimy>
DEVICE static inline void _process_emitting_tokens(processTokens_params* params) {
  // blockDim threads per token to process out-arcs in parallel
  auto group = cooperative_groups::tiled_partition<blockDimx>
               (cooperative_groups::this_thread_block());
  CostType cutoff = *params->cutoff;
  int32 size = params->prev_toks.Size();

  while (true) {
    int32 i;
    // We use a dispatcher in charge of
    // global scheduling, and make N threads as a group (N = 32)
    // to process all arcs from a single token. When the token is
    // processed, the group requests from the dispatcher a new token.
    // We implement task dispatching as an atomic operation.
    if (group.thread_rank() == 0) { // thread 0 nominated to get new token
      i = atomicAdd(params->pe_idx, 1); // allocate new token index
    }
    i = group.shfl(i, 0); // rank 0 broadcasts i to whole group
    if (i >= size) break; // finish processing all tokens

    TokenState& ts = params->prev_toks[i];
    Token * tok = params->lattice_processor.GetTokenByExactIdx(ts.tok_idx_allocated);

    if (size > params->max_active && tok->cost_ > *params->cutoff_prev)
      continue; // histogram pruning for last frame

    StateId state = ts.state;
    uint32 start = params->e_offsets[state], finish = params->e_offsets[state + 1];
    int32 ilabel, ilabel_next;  // prefetch ilabel since it leads to a dependent load
    int32 j = start + group.thread_rank();
    if (j < finish) {
      ilabel_next = params->arc_ilabels[j];
    }
    int32 nextj;

    for (j; j < finish; j = nextj) { // thread parallelism
      nextj = j + blockDimx;
      ilabel = ilabel_next;

      if (nextj < finish) {
        // prefetch ilabel since it leads to a dependent load
        ilabel_next = params->arc_ilabels[nextj];
      }
      BaseFloat acoustic_cost = -params->loglikelihoods[ilabel];
      BaseFloat weight = params->arc_weights[j];
      StateId nextstate = params->arc_nextstates[j];
      CostType total_cost = tok->cost_ + weight + acoustic_cost;

      if (total_cost <= cutoff) { // not prune out
        uint64* token_pack;
        TokenState *next_ts = NULL;
        int32 update_idx;
        // get cur_tok&token_pack addr
        _find_or_add_token_arc(params, nextstate, total_cost,
                              acoustic_cost, &ts, j, true, &next_ts, &token_pack,
                              &update_idx, true);
        // 1st stage of 2-pass atomic token recombination
        // get cur_te&new_token_pack here
        // details in the definition of pack_cost_idx_into_uint64()
        uint64 new_token_pack = pack_cost_idx_into_uint64(-total_cost, update_idx);
        uint64 ret = atomicMax((unsigned long long int *)token_pack,
                               (unsigned long long int)new_token_pack);
        if (ret < new_token_pack) {
          Token* cur_te = params->token_per_arc + update_idx;
          fast_store8(cur_te, &(Token(acoustic_cost + weight, tok)));
          params->token_per_arc_update[update_idx] = 1;
        }
      } // end total_cost<=cutoff
    } // end arc loop
  } // end token loop
  grid_sync(params->barrier); // after finishing all tokens
  // 2nd stage of 2-pass atomic token recombination
  params->cur_toks.StoreDataByPackIdx(params->token_per_arc,
                                      params->token_per_arc_update, params->numArcs,
                                      &(params->lattice_processor));
}

template<int32 blockDimx, int32 blockDimy>
DEVICE static inline void _process_nonemitting_tokens(processTokens_params
    *params, CostType cutoff, uint32 size,  volatile int32 *modified,
    bool aggregate = false) {
  assert(size);
  // blockDim threads per token to process out-arcs in parallel
  auto group = cooperative_groups::tiled_partition<blockDimx>
               (cooperative_groups::this_thread_block());

  // as we can't use linked list in GPU, a naive implementation is processing
  // all tokens in each iteration. We propose a better method by firstly doing
  // an aggregation of the Tokens updated in last iteration. After that,
  // processing non-emitting tokens is only conducted on these aggregated
  // tokens. For the first time to run into this iteration, all tokens are
  // updated by _process_emitting_tokens(), so we don't need to do aggregation
  // TODO: reduce number of iterations in processing non-emitting tokens
  int* agg_tok_idx = params->agg_idx; // need to make it 0 before enter this func
  int* cur_tok_idx = params->ne_idx; // need to make it 0 before enter this func
  int32 tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (aggregate) {  // aggregation stage
    for (tid; tid < size; tid += blockDim.x * gridDim.x) {
      if (params->cur_toks.IsUpdated(tid)) {
        int32 i = atomicAdd(agg_tok_idx, 1);
        // get updated token index for faster processing non-emitting tokens
        if (i >= size) break;
        params->ne_queue[i] = tid;
      }
    }
    grid_sync(params->barrier);
  }
  if (params->verbose > 3 && threadIdx.x == 0
      && blockIdx.x == 0) CUDA_PRINTF("PNE: %i %i %i %f\n", params->frame,
                                        params->cur_toks.Size(), *agg_tok_idx, *params->cutoff);

  while (true) {
    int32 i, j;
    // uses dynamically load balanced loop trips.
    // details are described in _process_emitting_tokens()
    if (group.thread_rank() == 0) { // thread 0 nominated to get new token
      if (aggregate) {
        j = atomicAdd(cur_tok_idx, 1); // allocate new token index
        if (j >= *agg_tok_idx) i = size; // to finish
        else i = params->ne_queue[j];
      } else {
        i = atomicAdd(cur_tok_idx, 1);
      }
    }
    i = group.shfl(i, 0); // rank 0 broadcasts i to whole group
    if (i >= size) break;

    TokenState& ts = params->cur_toks[i];
    Token * tok = params->lattice_processor.GetTokenByExactIdx(ts.tok_idx_allocated);
    StateId state = ts.state;
    assert(params->ne_offsets);
    uint32 start = params->ne_offsets[state], finish = params->ne_offsets[state + 1];
    for (int32 j = start + group.thread_rank(); j < finish; j += blockDimx) {
      BaseFloat weight = params->arc_weights[j];
      StateId nextstate = params->arc_nextstates[j];
      Token next_tok = Token(weight, tok);
      CostType total_cost = tok->cost_ + weight;

      // 2-pass atomic based token recombination
      // details described in _process_emitting_tokens()
      if (next_tok.cost_ <= cutoff) {
        TokenState *next_ts = NULL;
        uint64* token_pack;
        int32 update_idx;
        _find_or_add_token_arc(params, nextstate, total_cost,
                              0, &ts, j, true, &next_ts, &token_pack,
                              &update_idx, false);
        uint64 new_token_pack = pack_cost_idx_into_uint64(-total_cost, update_idx);
        uint64 ret = atomicMax((unsigned long long int *)token_pack,
                               (unsigned long long int)new_token_pack);
        if (ret < new_token_pack) {
          assert(update_idx < params->max_lat_arc_per_frame);
          Token* cur_te = params->token_per_arc + update_idx;
          fast_store8(cur_te, &(Token(weight, tok)));
          params->token_per_arc_update[update_idx] = 1;
          (*modified) = true; // show that we need another iteration
        }
      }
    }
  }
  grid_sync(params->barrier);
  params->cur_toks.StoreDataByPackIdx(params->token_per_arc,
                                      params->token_per_arc_update, params->numArcs,
                                      &(params->lattice_processor));
}

// end of inline device functions

// cuda __global__ functions

// before token passing, initialize lookuptables of all states 
// to prepare for the lookup
__global__
static void _initialize_all_states(TokenLookupElem *current_tokens_lookup,
                                 int32 numStates, int32 *barrier) {
  for (int32 i = blockIdx.x * blockDim.x + threadIdx.x; i < numStates;
       i += blockDim.x * gridDim.x) {
    current_tokens_lookup[i].tokenstate_idx = LOOKUP_DEACTIVE;
    // do not need to clear Token* cur_tok, as data will be re-stored in it
    // in StoreDataByPackIdx()
  }
  grid_sync(barrier);
  // we do not allocate token, so do not need this
  // if (blockIdx.x == 0 && threadIdx.x == 0) allocator.AdvanceFront(numStates);
}

// initialize by add the first token in the start state of WFST
__global__
static void _add_initial_token(processTokens_params params, StateId state) {
  TokenState *next_ts = NULL;
  uint64* token_pack;
  int32 j = 0;
  int32 update_idx = 0;
  if (threadIdx.x != 0 || blockIdx.x != 0) return;
  
  // putting this here to avoid extra kernel launch cost
  _initialize_cutoff(params.cutoff);

  _find_or_add_token_arc(&params, state, 0, // add first token
                        0, NULL, j, false,  &next_ts,
                        &token_pack, &update_idx, false);
  uint64 new_token_pack = pack_cost_idx_into_uint64(0, update_idx);
  Token* cur_te = params.token_per_arc + update_idx;
  params.token_per_arc_update[update_idx] = 1;
  fast_store8(cur_te, &(Token(0, NULL)));
  atomicMax((unsigned long long int *)token_pack, (unsigned long long int)new_token_pack);
  params.cur_toks.StoreDataByPackIdx(params.token_per_arc,
                                     params.token_per_arc_update, params.numArcs,
                                     &(params.lattice_processor));
}

// providing additional information of (maxThreadsPerBlock, minBlocksPerMultiprocessor)
// to the compiler to make more threads and blocks reside on GPU
__launch_bounds__(64, 64)
__global__
static void _process_tokens(processTokens_params params, bool is_init = false) {
  bool rank0 = blockIdx.x == 0 && threadIdx.x == 0;
  if (!is_init) { // only do _process_nonemitting_tokens() at frame 0
    _find_best_cutoff<32, 2>(&params);
    grid_sync(params.barrier);

    // to reduce a grid sync, we initialize here for next frame
    if (rank0)
      params.histogram_prev_toks.Initialize(*params.cutoff - params.beam);
  } else if (rank0) {
    *params.num_arcs_till_last = 0;
  }

  // modified flag for current iteration used in _process_nonemitting_tokens()
  volatile int32 *modified0 = params.modified;
  // modified flag for next/last iteration
  volatile int32 *modified1 = params.modified + 1;
  *modified1 = false;
  CostType cutoff = *params.cutoff;

  if (rank0) {
    *modified0 = false;
    *modified1 = false;
  }
  if (!is_init) { // only do _process_nonemitting_tokens() at frame 0
    _process_emitting_tokens<32, 2>(&params);
    grid_sync(params.barrier);  // ensure cur_toks size is final
  }

  // debug
  int32 tok_E;
  int32 itv = params.verbose > 2 ? 1 : 10;
  // if without -G, this code will be optimized out
  if (rank0 && params.verbose > 0 && params.frame % itv == 0)
    tok_E = params.cur_toks.Size();
  int32 cnt = 0;
  uint32 size = 0;
  do {
    size = params.cur_toks.Size();
    if (rank0) {
      // need to make it 0 before enter _process_nonemitting_tokens
      *params.ne_idx = 0;
      // need to make it 0 before enter _process_nonemitting_tokens
      *params.agg_idx = 0;
    }
    // wait for everyone to read size and modified0
    grid_sync(params.barrier);

    // swap buffers: double buffered to avoid extra sync when resetting
    // modified to false, 3% speedup
    // if we use more modified, we can reduce more grid sync,
    // but will make the program complexer
    cuda_swap(modified0, modified1);
    if (rank0) *modified1 = false;
    cnt++;
    // details of aggregation described in _process_nonemitting_tokens()
    bool aggregate = (!is_init) && cnt > 1 ? 1 : 0;
    _process_nonemitting_tokens<32, 2>(&params, cutoff, size, modified0, aggregate);

    // we have sync in the end of _process_nonemitting_tokens
    // grid_sync(params.barrier);
    // wait for everyone to finish process tokens and writes modified0
  } while ((*modified0) == true && cnt < 10);
  if (rank0 && params.verbose > 0 && params.frame % itv == 0)
    CUDA_PRINTF("TK: %i %i %i %f\n", params.frame, tok_E,
                params.cur_toks.Size(), cutoff);

  // process lattice before allocate new toks to TokenState
  params.lattice_processor.CollectToksPerFrame(params.cur_toks, params.frame);
  // accumulatively store lattice arcs
  params.lattice_processor.CollectArcsPerFrame(params.lat_arcs_sub_vec,
      params.frame);
  if (rank0) {
    *params.num_arcs_till_last = params.lat_arcs_sub_vec.Size();
  }

  grid_sync(params.barrier); // after process lattice

  _initialize_visited_states(params.current_tokens_lookup, params.cur_toks);

  if (rank0) {
    // prepare for next iteration
    *params.cutoff = INFINITY;
    *params.fb_idx = 0;
    *params.pe_idx = 0;
  }
}

// providing additional information of (maxThreadsPerBlock, minBlocksPerMultiprocessor)
// to the compiler to make more threads and blocks reside on GPU
__launch_bounds__(64, 64)
__global__
static void _prune_active_tokens(processTokens_params params) {
  params.lattice_processor.PruneActiveTokens(params.frame, params.lattice_beam,
                                          params.verbose);
}

// end of cuda __global__ functions

// CudaVector Implementation
template<typename T>
void CudaVector<T>::Allocate(uint32 max_size,
                                    uint32* count_h, uint32* count_d, T* mem_h, T* mem_d) {
  alloc_size = 0;
  this->max_size = max_size;

  if (count_h) this->count_h = count_h;
  else hipHostMalloc(&this->count_h, sizeof(uint32));
  if (count_d) this->count_d = count_d;
  else {
    alloc_size += sizeof(uint32);
    hipMalloc(&this->count_d, sizeof(uint32));
  }
  if (mem_h) this->mem_h = mem_h;
  else hipHostMalloc(&this->mem_h, max_size * sizeof(T));
  if (mem_d) this->mem_d = mem_d;
  else {
    alloc_size += max_size * sizeof(T);
    hipMalloc(&this->mem_d, max_size * sizeof(T));
  }

  hipMemset(this->count_d, 0, sizeof(uint32));
  *this->count_h = 0;
}

template<typename T>
void CudaVector<T>::Free(bool create_outside) {
  hipHostFree(mem_h);
  if (!create_outside) {
    hipFree(mem_d);
  }
  hipHostFree(count_h);
  hipFree(count_d);
}

template<typename T>
HOST DEVICE T& CudaVector<T>::operator[](uint32 idx) {
#ifdef __CUDA_ARCH__
  assert(idx < *count_d);
  return mem_d[idx];
#else
  assert(idx < *count_h);
  return mem_h[idx];
#endif
}

template<typename T>
HOST DEVICE const T& CudaVector<T>::operator[](uint32 idx) const {
#ifdef __CUDA_ARCH__
  assert(idx < *count_d);
  return mem_d[idx];
#else
  assert(idx < *count_h);
  return mem_h[idx];
#endif
}

// This will cause page faults back and forth when we switch from host to device.
// need to call e.g. CopySizeToHost() before this function
template<typename T>
HOST DEVICE uint32 CudaVector<T>::Size() const {
#ifdef __CUDA_ARCH__
  return *count_d;
#else
  return *count_h;
#endif
}

// push back function implemented
// by an atomic operation, where the memory is pre-allocated
template<typename T>
HOST DEVICE uint32 CudaVector<T>::PushBack(const T &val) {
#ifdef __CUDA_ARCH__
  uint32 idx = atomicAdd(count_d, 1);
#ifdef __DEBUG__
  assert(*count_d < max_size);
#else
  if (*count_d >= max_size) *count_d = max_size - 1;
#endif
  if (sizeof(T) == 16) fast_store16(mem_d + idx, &val);
  else mem_d[idx] = val;
#else
  assert(*count_h < max_size);
  uint32 idx = (*count_h)++;
  mem_h[idx] = val;
#endif
  return idx;
}

template<typename T>
HOST DEVICE void CudaVector<T>::Clear(hipStream_t stream) {
#ifdef __CUDA_ARCH__
  *count_d = 0;
#else
  *count_h = 0;
  hipMemsetAsync(count_d, 0, sizeof(int32), stream);
#endif
}

template<typename T>
void CudaVector<T>::Swap(CudaVector<T> &v) {
  std::swap(mem_h, v.mem_h);
  std::swap(mem_d, v.mem_d);
  std::swap(count_h, v.count_h);
  std::swap(count_d, v.count_d);
  std::swap(max_size, v.max_size);
}

// given an allocated address in vector memory, calculate its index in the vector
template<typename T>
HOST DEVICE int32 CudaVector<T>::GetIdxFromAddr(T* addr) {
#ifdef __CUDA_ARCH__
  int32 ret = addr - mem_d;
  assert(ret < *count_d && ret >= 0);
  return ret;
#else
  int32 ret = addr - mem_h;
  assert(ret < *count_h && ret >= 0);
  return ret;
#endif
}

// a series of data transfer functions between host and device
template<typename T>
void CudaVector<T>::CopyAllToHost(hipStream_t stream) {
  hipStreamSynchronize(stream);
  hipMemcpy(count_h, count_d, sizeof(int32), hipMemcpyDeviceToHost);
  hipMemcpyAsync(mem_h, mem_d, *count_h * sizeof(T), hipMemcpyDeviceToHost,
                  stream);
}

template<typename T>
void CudaVector<T>::CopyAllToDevice(hipStream_t stream) {
  hipStreamSynchronize(stream);
  hipMemcpyAsync(count_d, count_h, sizeof(int32), hipMemcpyHostToDevice);
  hipMemcpyAsync(mem_d, mem_h, *count_h * sizeof(T), hipMemcpyHostToDevice,
                  stream);
}

template<typename T>
void CudaVector<T>::CopySizeToHost(hipStream_t stream) {
  hipMemcpyAsync(count_h, count_d, sizeof(int32), hipMemcpyDeviceToHost, stream);
}

template<typename T>
void CudaVector<T>::CopySizeToDevice(hipStream_t stream) {
  hipMemcpyAsync(count_d, count_h, sizeof(int32), hipMemcpyHostToDevice, stream);
}

template<typename T>
void CudaVector<T>::CopyDataToHost(hipStream_t stream, T* to_buf,
    bool copy_size) {
  if (!to_buf) {
    to_buf = mem_h;
  }
  if (copy_size) hipMemcpy(count_h, count_d, sizeof(int32),
                              hipMemcpyDeviceToHost);
  hipMemcpyAsync(to_buf, mem_d, *count_h * sizeof(T), hipMemcpyDeviceToHost,
                  stream);
}

template<typename T>
void CudaVector<T>::CopyDataToDevice(hipStream_t stream) {
  hipMemcpyAsync(mem_d, mem_h, *count_h * sizeof(T), hipMemcpyHostToDevice,
                  stream);
}

// CudaVector Implementation

template<typename T>
void CudaMergeVector<T>::Allocate(uint32 max_size) {
  CudaVector<T>::Allocate(max_size);

  hipMalloc(&mem_update_d, sizeof(int32) * max_size);
  hipMalloc(&barrier_, sizeof(int32) * 1);

  hipMemset(mem_update_d, 0, sizeof(int32) * max_size);
}

template<typename T>
void CudaMergeVector<T>::Free() {
  CudaVector<T>::Free();

  hipFree(mem_update_d);
  hipFree(barrier_);
}

template<typename T>
void CudaMergeVector<T>::Swap(CudaMergeVector<T> &v) {
  CudaVector<T>::Swap(v);
  std::swap(mem_update_d, v.mem_update_d);
}

template<typename T>
size_t CudaMergeVector<T>::GetCudaMallocBytes() {
  return CudaVector<T>::GetCudaMallocBytes() +
         sizeof(uint32) * (1 + 2 * (2)) + max_size * (sizeof(T) +
             sizeof(uint64*) + sizeof(int32));
}


template<typename T>
DEVICE void CudaMergeVector<T>::StoreDataByPackIdx(
  void* temp_data_buf, int* temp_data_buf_update, int32 buf_size,
  LatticeProcessor* lattice_processor) {
  assert(0);  // haven't implemented
}

// according to the unpack index, copy data from external buf to the inside
// buf; it's used in the 2nd stage of 2-pass atomic token recombination
// Namely, in each frame, we save the token
// information in an array whose size is the number of arcs. This
// ensures there are no write conflicts between threads since each
// arc can be accessed at most once in each frame. After passing
// all tokens, we aggregate survived packed tokens, unpack them
// to get arc indexes, and store token information from the former
// array to token data structures exploiting thread parallelism.
template<>
DEVICE void CudaMergeVector<TokenState>::StoreDataByPackIdx(
  void* temp_data_buf, int* temp_data_buf_update, int32 buf_size,
  LatticeProcessor* lattice_processor) {
  int32 tid = threadIdx.x + blockIdx.x * blockDim.x;
  int32 batch = blockDim.x * gridDim.x;
  int32 size = *count_d; // count_d is cleared in Clear() called by InitDecoding()

  for (; tid < size; tid += batch) { // thread parallelism
    uint64* pack_v = &mem_d[tid].token_pack;
    int32 idx = unpack_idx_from_uint64(*pack_v);
    assert(idx < buf_size);
    mem_update_d[(tid + 0)] = temp_data_buf_update[idx];
    if (temp_data_buf_update[idx]) temp_data_buf_update[idx] = 0;
    else continue; // if it isn't updated, just skip storing
    TokenState* to_ts = mem_d + (tid + 0);
    Token* cur_tok = ((Token *)temp_data_buf) + idx;
    Token* to_tok = lattice_processor->GetTokenByExactIdx(to_ts->tok_idx_allocated);
    fast_store8(to_tok, cur_tok); // memcpy(to_tok,cur_tok,sizeof(T));
  }
}

// check whether the item in index i of the vector is updated in this frame
// call this function after StoreDataByPackIdx()
template<typename T>
DEVICE int32 CudaMergeVector<T>::IsUpdated(int32 i) {
  if (i >= *count_d) return 0;
  return mem_update_d[i];
}

// push back function implemented
// by an atomic operation, where the memory is pre-allocated
template<typename T>
DEVICE uint32 CudaMergeVector<T>::PushBack(const T &val,
    uint64 *val_pack) {
  assert(0); //this func is deprecated
  uint32 idx = atomicAdd(count_d, 1);
  assert(*count_d < max_size);
  assert(sizeof(val) == 16); // use faster storing
  fast_store16(&mem_d[idx], &val);
  // store the pack_data pointer in 1st stage
  // mem_pack_buf_d[idx] = val_pack; // used in StoreDataByPackIdx() in 2nd stage, it's always stored in mem_d now
  return idx;
}

// LatticeProcessor Implementation
// Initialize in InitDecoding()
void LatticeProcessor::Initialize() {
  hipMemset(arcs_apr_fr_size_d, 0, sizeof(int32) * (prune_interval + 2));
  hipMemset(arcs_apr_used_d, 0, sizeof(int32));
  hipMemset(arcs_bpr_used_d, 0, sizeof(int32));
  hipMemset(toks_bpr_fr_sidx_d, 0, sizeof(int32) * (prune_interval + 2));
  hipMemset(arcs_bpr_fr_sidx_d, 0, sizeof(int32) * (prune_interval + 2));
  hipMemset(toks_num_used, 0, sizeof(int32));
}

// the return value including the hipMallocManaged size
int32 LatticeProcessor::Allocate(int32 max_tokens_per_frame,
                              int32 max_lat_arc_per_frame, int32 prune_interval,
                              int32 max_toks, int32 max_arcs,
                              const CudaFst& fst) {
  int32 sz;
  int32 bytes_cuda_malloc = 0;

  // before pruning
  // to reduce memory usage, we use hipMallocManaged, which doesn't
  // allocate in GPU at once
  sz = sizeof(Token) * max_toks;
  cuda_malloc_managed_preferred_device((void**)&toks_bpr_d, sz);
  bytes_cuda_malloc += sz;
  // if we directly use managed memory from toks_bpr_d, the RTF is 10% larger
  hipHostMalloc((void**)&toks_bpr_h, sz);
  toks_buf_before_pr_size = sz / sizeof(Token);

  // to reduce memory usage, we use hipMallocManaged, which doesn't
  // allocate in GPU at once
  sz = sizeof(LatLinkCompact) * max_arcs;
  cuda_malloc_managed_preferred_device((void**)&arcs_bpr_d, sz);
  bytes_cuda_malloc += sz;

  arcs_buf_before_pr_size = max_arcs;
  sz = sizeof(int32) * (prune_interval + 2);
  hipMalloc((void**)&toks_bpr_fr_sidx_d, sz); bytes_cuda_malloc += sz;
  hipHostMalloc((void**)&toks_bpr_fr_sidx_h, sz);
  sz = sizeof(int32);
  hipMalloc((void**)&toks_num_used, sz); bytes_cuda_malloc += sz;
  sz = sizeof(int32) * (prune_interval + 2);
  hipMalloc((void**)&arcs_bpr_fr_sidx_d, sz); bytes_cuda_malloc += sz;

  // after pruning
  sz = sizeof(int32) * (prune_interval + 2);
  hipMalloc((void**)&arcs_apr_fr_size_d, sz); bytes_cuda_malloc += sz;
  hipHostMalloc((void**)&arcs_apr_fr_size_h, sz);
  sz = ESTIMATED_PRUNE_RATIO * sizeof(LatLink) * max_arcs;
  // to reduce memory usage, we use hipMallocManaged, which doesn't
  // allocate in GPU at once
  cuda_malloc_managed_preferred_device((void**)&arcs_apr_d, sz);
  bytes_cuda_malloc += sz;
  hipHostMalloc((void**)&arcs_apr_h, sz);
  sz = sizeof(int32);
  hipMalloc((void**)&arcs_apr_used_d, sz); bytes_cuda_malloc += sz;
  hipMalloc((void**)&arcs_bpr_used_d, sz); bytes_cuda_malloc += sz;
  hipHostMalloc((void**)&arcs_apr_used_h, sz);

  // GPU global memory temp variables
  sz = sizeof(int32);
  hipMalloc((void**)&barrier_, sz); bytes_cuda_malloc += sz;
  sz = sizeof(int32) * 3;
  hipMalloc((void**)&modified_d, sz); bytes_cuda_malloc += sz;
  sz = sizeof(int32) * (2);
  hipMalloc((void**)&count_vec_acc_d, sz); bytes_cuda_malloc += sz;
  this->prune_interval = prune_interval;

  arc_ilabels = fst.arc_ilabels_d;
  arc_olabels = fst.arc_olabels_d;
  arc_weights = fst.arc_weights_d;
  return bytes_cuda_malloc;
}
void LatticeProcessor::Free() {
  // before pruning
  hipFree(arcs_bpr_used_d);
  hipHostFree(arcs_apr_used_h);
  //hipFree(toks_bpr_d);
  hipHostFree(toks_bpr_h);
  hipFree(arcs_bpr_d);
  hipFree(toks_bpr_fr_sidx_d);
  hipHostFree(toks_bpr_fr_sidx_h);
  hipFree(arcs_bpr_fr_sidx_d);
  hipFree(toks_num_used);

  // after pruning
  hipFree(arcs_apr_fr_size_d);
  hipHostFree(arcs_apr_fr_size_h);
  hipFree(arcs_apr_d);
  hipFree(arcs_apr_used_d);

  // GPU global memory temp variables
  hipFree(count_vec_acc_d);
  hipFree(barrier_);
  hipFree(modified_d);
  hipHostFree(arcs_apr_h);
}

DEVICE Token* LatticeProcessor::GetTokenByExactIdx(uint32 offset) {
  int32 idx = offset;
#ifdef __DEBUG__
  assert(idx >= 0 && idx < toks_buf_before_pr_size);
#else
  if (idx >= toks_buf_before_pr_size) idx = toks_buf_before_pr_size - 1;
#endif
  return toks_bpr_d + idx;
}

DEVICE int32 LatticeProcessor::GetTokenAllocIdx(uint32 offset) {
  int32 idx = *toks_num_used + offset;
#ifdef __DEBUG__
  assert(idx >= 0 && idx < toks_buf_before_pr_size);
#else
  if (idx >= toks_buf_before_pr_size) idx = toks_buf_before_pr_size - 1;
#endif
  return idx;
}

DEVICE int32 LatticeProcessor::GetTokenIdxFromAddr(Token* tok) {
  int32 ret = tok - toks_bpr_d;
  assert(ret < toks_buf_before_pr_size && ret >= 0);
  return ret;
}

// entry of lattice pruning until this frame
DEVICE void LatticeProcessor::PruneActiveTokens(int32 frame,
    BaseFloat lattice_beam, int32 verbose) {
  int32 rank0 = threadIdx.x == 0 && blockIdx.x == 0 ? 1 : 0;
  if (frame == 0) return;
  if (rank0) *arcs_apr_used_d = 0; // clear buffer index
  grid_sync(barrier_);
  for (int32 f = frame; f > 0; f--) { // prune each frame in serial
    PruneLatticeForFrame(f, 1, lattice_beam, verbose);
  }
  // by ESTIMATED_PRUNE_RATIO to reduce memory allocation and D2H data transfer
  assert(*arcs_apr_used_d < arcs_buf_before_pr_size * ESTIMATED_PRUNE_RATIO);
  if (verbose > 2 && rank0)
    CUDA_PRINTF("PRt: %i %i\n", arcs_bpr_fr_sidx_d[frame + 1],
                *arcs_apr_used_d);
}

// collect after each token passing, we store Token data in the sequence of
// TokenState vector, using continuous memory
DEVICE void LatticeProcessor::CollectToksPerFrame(
  TokenMergeVector& cur_toks_vec, int32 frame) {
  int32 tid = threadIdx.x + blockIdx.x * blockDim.x;
  TokenState* cur_toks = cur_toks_vec.mem_d;
  int32 size = cur_toks_vec.Size();
  if (tid == 0) {
    // Set start index in the buffer of the next frame
    SetNextSidx(toks_bpr_fr_sidx_d, size, frame);
    *toks_num_used += size;
    assert(*toks_bpr_fr_sidx_d < toks_buf_before_pr_size);
  }
}

// collect after each token passing, mainly to update arcs_bpr_fr_sidx_d here
DEVICE void LatticeProcessor::CollectArcsPerFrame(LatLinkVector&
    cur_arc_array,
    int32 frame) {
  int32 tid = threadIdx.x + blockIdx.x * blockDim.x;
  int32 idx = tid;
  int32 rank0 = blockIdx.x == 0 && threadIdx.x == 0 ? 1 : 0;
  int32 batch = blockDim.x * gridDim.x;

  int32 size = cur_arc_array.Size() - *arcs_bpr_used_d; // size of current frame
  grid_sync(barrier_);
  if (rank0) {
    SetNextSidx(arcs_bpr_fr_sidx_d, size, frame);
    *arcs_bpr_used_d = cur_arc_array.Size();
    // we didn't clear cur_arc_array.count_d until the end of decoding
  }
  /*
  // we share the memory between vector&pruner, so dont need to copy between them
  for(; idx < size; idx += batch) {
    LatLink* to_arc=GetActiveArc(frame,(idx));
    fast_store32(to_arc, cur_arc_array.mem_d+idx);
    // for debug purpose
    GetActiveToken((cur_arc_array.mem_d+idx)->p1,true,frame);
    GetActiveToken(to_arc->p1,true,frame);
  }
  */
}

// AddArc function implemented
// by an atomic operation, where the memory is pre-allocated
DEVICE int32 LatticeProcessor::AddArc(LatLink* arc) {
  int32 i = atomicAdd(arcs_apr_used_d, 1);
  assert(i < arcs_buf_before_pr_size * ESTIMATED_PRUNE_RATIO);
  fast_store32(arcs_apr_d + i, arc);
}
DEVICE int32 LatticeProcessor::AddArc(LatLinkCompact* arc, int32 frame) {
  int32 i = atomicAdd(arcs_apr_used_d, 1);
  assert(i < arcs_buf_before_pr_size * ESTIMATED_PRUNE_RATIO);
  int32 frame_tok = arc->IsEmitArc() ? frame - 1 : frame;
  int32 j = arc->arc_id;
  LatLink apr_arc(arc->GetPrevTokId(), frame_tok, arc->next_tok_id, frame,
                  arc_ilabels[j], arc_olabels[j], arc_weights[j], arc->acoustic_cost);
  fast_store32(arcs_apr_d + i, &apr_arc);
}


// Set start index in the buffer of the next frame
DEVICE void LatticeProcessor::SetNextSidx(int* sidx_buf, int32 size,
    int32 frame) {
  assert(frame >= 0);
  int32 cur_sidx = sidx_buf[(frame)];
  sidx_buf[(frame + 1)] = cur_sidx + size;
}

// Get the active token indexed by a uint64 pair (frame, idx), stored in void* p
// the details of the pair can be referred to LatLink::LatLink()
DEVICE Token* LatticeProcessor::GetActiveToken(void* p, bool check,
    int32 iframe) const {
  int32 frame, id;
  DECODE_TOK_IDX_PAIR(frame, id, (uint64)p);
  if (check) assert(frame == iframe || frame == iframe - 1);
  return GetActiveToken(frame, id, check);
}

// Get the active token indexed by a uint64 pair (frame, idx)
// the details of the pair can be referred to LatLink::LatLink()
DEVICE Token* LatticeProcessor::GetActiveToken(int32 frame, int32 id_pack,
    bool check) const {

  int32 cur_sidx = toks_bpr_fr_sidx_d[frame];
  int32 id = id_pack & ((1 << 31) - 1);
  assert(cur_sidx + id < toks_buf_before_pr_size);
  Token* tok = toks_bpr_d + cur_sidx + id;
  /*
  if (check) {
    assert(tok->frame == frame);
  }
  */
  return tok;
}

// Get the active token indexed by a uint64 pair (frame, idx)
// the details of the pair can be referred to LatLink::LatLink()
DEVICE Token* LatticeProcessor::GetActiveTokenByExactId(int32 frame,
    int32 id_exact, bool check) const {
  Token* tok = toks_bpr_d + id_exact;

  if (check) {
    if (id_exact < toks_bpr_fr_sidx_d[frame]) CUDA_PRINTF("h %i %i\n", id_exact,
          toks_bpr_fr_sidx_d[frame]);
    if (id_exact >= toks_bpr_fr_sidx_d[frame + 1]) CUDA_PRINTF("t %i %i\n", id_exact,
          toks_bpr_fr_sidx_d[frame + 1]);
    assert(toks_bpr_fr_sidx_d[frame] <= id_exact &&
           id_exact < toks_bpr_fr_sidx_d[frame + 1]);
  }

  return tok;
}

// Get the active arc indexed by a uint64 pair (frame, idx)
// the vector memory and the start index of each frame are kept in LatticeProcessor
DEVICE LatLinkCompact* LatticeProcessor::GetActiveArc(int32 frame,
    int32 id) const {
  int32 cur_sidx = arcs_bpr_fr_sidx_d[(frame)];
  assert(cur_sidx + id < arcs_buf_before_pr_size);
  LatLinkCompact* arc = arcs_bpr_d + cur_sidx + id;
  return arc;
}

// Size of items in the frame, it is obtained from an accumulate number array
DEVICE int32 LatticeProcessor::GetSize(int* acc_len, int32 frame) const {
  int32 size = acc_len[(frame) + 1] - acc_len[(frame)];
  assert(size >= 0 && size <= arcs_buf_before_pr_size);
  return size;
}

// used in PruneLatticeForFrame()
DEVICE void LatticeProcessor::UpdateModifiedFlags(
  volatile int32 **modified0, volatile int32 **modified1,
  volatile int32 **modified2, int cnt, int32 *modified_d) {
  *modified0 = modified_d + cnt % 3;
  *modified1 = modified_d + (cnt + 1) % 3;
  *modified2 = modified_d + (cnt + 2) % 3;
}

// The parallel lattice pruning is based on the algorithm in
// LatticeFasterDecoder::PruneActiveTokens
// with necessary modifications for GPU parallelization:
// i) parallelize the iterative updating of nodes and arcs over GPU
// threads; ii) use a global arc vector to replace the linked lists in
// the old implementation, for its lack of random access features to
// enable parallel access; iii) implement the extra cost updating as
// an atomic operation to eliminate write conflicts among threads.
// When a lattice arc is pruned, we do not physically remove
// the arc, as memory allocation is expensive. Instead, we do a
// final merging step to aggregate all remaining arcs using thread
// parallelism
// We do not prune lattice nodes because: i) we need a static mapping
// for each arc to trace the previous and the next nodes before
// and after D2H memory copy. We use frame index t and vector
// index i to trace a node, thus node positions in the vector cannot
// be changed. ii) the lattice is constructed in CPU by iterating
// remaining arcs, thus nodes are implicitly pruned. iii) node D2H
// copy is done in each frame asynchronously, which does not introduce overheads.
DEVICE void LatticeProcessor::PruneLatticeForFrame(int32 frame,
    bool merge, BaseFloat lattice_beam, int32 verbose) {
  int32 prev_cidx;
  int32 c = 0;
  int32 rank0 = threadIdx.x == 0 && blockIdx.x == 0 ? 1 : 0;
  volatile int32 *modified0;
  volatile int32 *modified1;
  volatile int32 *modified2;
  int32 cnt = 0;
  UpdateModifiedFlags(&modified0, &modified1, &modified2, cnt, modified_d);
  if (rank0 && verbose > 3) CUDA_PRINTF("%i %i\n", c++, GetSize(toks_bpr_fr_sidx_d,
                                          frame - 1)); // size before pruning
  {
    // initialize
    int32 tid = threadIdx.x + blockIdx.x * blockDim.x;
    int32 size = GetSize(toks_bpr_fr_sidx_d, frame - 1);
    for (; tid < size; tid += gridDim.x * blockDim.x) {
      Token* tok = GetActiveToken(frame - 1, tid, true);
      tok->extra_cost = FLT_MAX;
    }
    if (rank0) {
      *modified0 = 1;
      *modified1 = 0;
      *modified2 = 0;
      prev_cidx = *arcs_apr_used_d;
    }
    // wait for i) last iteration(frame+1) finish ii) finish initialization
    grid_sync(barrier_);
  }

  // iteratively updates extra costs of nodes and arcs until they stop changing,
  while (cnt++ < 10 && *modified0 != 0) {
    // triple buffer to eliminate a grid sync after *modified1 = 0;
    UpdateModifiedFlags(&modified0, &modified1, &modified2, cnt, modified_d);
    // till now, threads are using modified0 & modified2, so we clear
    // *modified1 here as it won't be used before grid sync in the very below
    if (rank0) *modified1 = 0;
    // wait for every thread to enter while, which slow down by 2% here
    //grid_sync(barrier_);

    int32 tid = threadIdx.x + blockIdx.x * blockDim.x;
    int32 size = GetSize(arcs_bpr_fr_sidx_d, frame);
    for (; tid < size; tid += gridDim.x * blockDim.x) {
      LatLinkCompact* link = GetActiveArc(frame, tid);
      int32 frame_tok = link->IsEmitArc() ? frame - 1 : frame;
      Token* next_tok = GetActiveToken(frame, link->next_tok_id, true);
      Token* tok = GetActiveToken(frame_tok, link->GetPrevTokId(), true);
      // extra cost is defined as the difference between the best
      // cost including the current arc and the best overall path.
      BaseFloat link_extra_cost = next_tok->extra_cost +
                                  ((tok->cost_ + link->acoustic_cost + arc_weights[link->arc_id])
                                   - next_tok->cost_);
      if (!isnan(link_extra_cost) && link_extra_cost <= lattice_beam) {
        // not prune out
        if (link_extra_cost < -1) {// debug
          CUDA_PRINTF("%i %f %f %f %f %f\n", frame, next_tok->extra_cost, tok->cost_,
                      link->acoustic_cost, arc_weights[link->arc_id], next_tok->cost_);
          link_extra_cost = lattice_beam / 2;
        }
        if (link_extra_cost < tok->extra_cost) {
          atomic_min(&tok->extra_cost, link_extra_cost);
          if (*modified0 == 0) atomicAdd((int32 *)modified0, 1);
        }
      }
    }
    grid_sync(barrier_);
    if (rank0 && verbose > 3) CUDA_PRINTF("%i %i\n", c++, cnt);
  }

  // final aggregate remaining arcs
  {
    int32 tid = threadIdx.x + blockIdx.x * blockDim.x;
    int32 size = GetSize(arcs_bpr_fr_sidx_d, frame);
    for (; tid < size; tid += gridDim.x * blockDim.x) {
      LatLinkCompact* link = GetActiveArc(frame, tid);
      int32 frame_tok = link->IsEmitArc() ? frame - 1 : frame;
      Token* next_tok = GetActiveToken(frame, link->next_tok_id, true);
      Token* tok = GetActiveToken(frame_tok, link->GetPrevTokId(), true);
      BaseFloat link_extra_cost = next_tok->extra_cost +
                                  ((tok->cost_ + link->acoustic_cost + arc_weights[link->arc_id])
                                   - next_tok->cost_);
      if (!isnan(link_extra_cost) && link_extra_cost <= lattice_beam) {
        // not pruned out
        if (merge) {
          AddArc(link, frame);
          // link->acoustic_cost=HIP_NAN_F;
          // don't need to delete it in original lattice
        }
      }
    }
    grid_sync(barrier_);
  }

  /*
  { // we do not prune lattice node
    // update tok
    int32 tid=threadIdx.x+blockIdx.x*blockDim.x;
    int32 size=GetSize(toks_bpr_fr_sidx_d,frame);
    for (;tid<size;tid+=gridDim.x*blockDim.x) {
      Token* tok=GetActiveToken(frame-1,tid);
      if (tok->extra_cost==FLT_MAX)
        tok->tot_cost=HIP_NAN_F; // prune
    }
  }
  */

  // get size
  if (merge && rank0) {
    int& size_arc_of_frame = arcs_apr_fr_size_d[frame];
    size_arc_of_frame = *arcs_apr_used_d - prev_cidx;
    if (verbose > 3 || (size_arc_of_frame == 0
                        && frame != 0)) CUDA_PRINTF("PR %i %i %i\n", frame,
                              GetSize(arcs_bpr_fr_sidx_d, frame), size_arc_of_frame);
  }
  // grid_sync(barrier_);
}

// copy accumulated arcs after lattice pruning till the given frame
// after obtaining the copy size, copy the buffer asynchronously
void LatticeProcessor::CopyArcsToHost(int32 frame, hipStream_t st) {
  int32 sz;
  hipMemcpy(arcs_apr_used_h, arcs_apr_used_d,
             sizeof(int32), hipMemcpyDeviceToHost);
  // TODO: optimize out above overhead
  // one possibility is we can copy static length
  // by assuming ESTIMATED_PRUNE_RATIO parts are remained
  // sz=sizeof(LatLink)*(arcs_buf_before_pr_size*ESTIMATED_PRUNE_RATIO);

  sz = sizeof(LatLink) * (*arcs_apr_used_h); // use exact count
  hipMemcpyAsync(arcs_apr_h, arcs_apr_d,
                  sz, hipMemcpyDeviceToHost, st);
  sz = sizeof(int32) * (frame + 1) * (1);
  hipMemcpyAsync(arcs_apr_fr_size_h, arcs_apr_fr_size_d,
                  sz, hipMemcpyDeviceToHost, st);
  // clear arcs_apr_used_d in GPU during next call of pruning
}

// copy accumulated toks till the given frame
// after obtaining the copy size, copy the buffer asynchronously
void LatticeProcessor::CopyToksToHost(int32 frame, hipStream_t st) {
  int32 sz;
  // include frame 0 count and the total count in the last element
  assert(frame <= prune_interval); // the max size of toks_bpr_fr_sidx_h
  sz = sizeof(int32) * (frame + 1 + 1) * (1);
  hipMemcpy(toks_bpr_fr_sidx_h, toks_bpr_fr_sidx_d,
             sz, hipMemcpyDeviceToHost);
  sz = sizeof(Token) * (toks_bpr_fr_sidx_h[frame + 1]);
  assert(sz); // assume we have obtain the total count
  hipMemcpyAsync(toks_bpr_h, toks_bpr_d,
                  sz, hipMemcpyDeviceToHost, st);
}

// get back the host data address which can be used in CPU lattice processing
void LatticeProcessor::GetHostData(Token** toks_buf, int** toks_fr_sidx,
                                LatLink** arcs_buf, int** arcs_fr_size) {
  *toks_fr_sidx = toks_bpr_fr_sidx_h;
  *toks_buf = toks_bpr_h;
  *arcs_fr_size = arcs_apr_fr_size_h; // prune_interval len
  *arcs_buf = arcs_apr_h; // start of prune_interval len arcs
}

// CudaLatticeDecoder Implementation
// constructor
CudaLatticeDecoder::CudaLatticeDecoder(const CudaFst &fst,
                                       const CudaLatticeDecoderConfig &config):
  config_(config), fst_(fst), bytes_cuda_malloc(0) {
  KALDI_VLOG(1) << "CudaLatticeDecoder Constructor\n";
  int32 device;
  hipGetDevice(&device);
  CU_SAFE_CALL(hipGetLastError());

  // for CUDA_PRINTF
  if (config_.verbose > 4) hipDeviceSetLimit(hipLimitPrintfFifoSize, 1e7);

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, device);

  // GPU utilization
  total_threads = prop.maxThreadsPerMultiProcessor * prop.multiProcessorCount *
                  config.gpu_fraction;

  bytes_cuda_malloc += histogram_prev_toks_.Allocate(config_.beam,
                       (int32)(config_.beam * 0.5), 1.0);

  hipEventCreateWithFlags(&event_pt, hipEventDisableTiming);
  hipEventCreateWithFlags(&event_ll, hipEventDisableTiming);

  hipStreamCreateWithFlags(&stream_comp, hipStreamNonBlocking);
  for (int32 i = 0; i < LAT_BUF_SIZE; i++)
    hipStreamCreateWithFlags(&stream_lat[i], hipStreamNonBlocking);
  hipStreamCreateWithPriority(&stream_ll, hipStreamNonBlocking, -1);

  hipMalloc(&pe_idx_d, sizeof(int32)); bytes_cuda_malloc += sizeof(int32);
  hipMalloc(&agg_idx_d, sizeof(int32)); bytes_cuda_malloc += sizeof(int32);
  hipMalloc(&ne_idx_d, sizeof(int32)); bytes_cuda_malloc += sizeof(int32);
  hipMalloc(&ne_queue_d, sizeof(int32)*config.max_tokens_per_frame);
  bytes_cuda_malloc += sizeof(int32);
  hipMalloc(&fb_idx_d, sizeof(int32)); bytes_cuda_malloc += sizeof(int32);
  hipMalloc(&barrier_d, sizeof(int32)); bytes_cuda_malloc += sizeof(int32);

  hipMemset(pe_idx_d, 0, sizeof(int32));
  hipMemset(ne_idx_d, 0, sizeof(int32));
  hipMemset(agg_idx_d, 0, sizeof(int32));
  hipMemset(fb_idx_d, 0, sizeof(int32));
  hipMemset(barrier_d, 0, sizeof(int32));
  CU_SAFE_CALL(hipGetLastError());

  hipMalloc(&cutoff_d, sizeof(CostType)); bytes_cuda_malloc += sizeof(CostType);
  hipMalloc(&cutoff_prev_d, sizeof(CostType));
  bytes_cuda_malloc += sizeof(CostType);
  hipMalloc((void**)&num_arcs_till_last_d, sizeof(int32));
  bytes_cuda_malloc += sizeof(int32);
  hipMalloc(&modified_d, sizeof(int32) * 2);
  bytes_cuda_malloc += sizeof(int32) * 2;

  hipMalloc((void**)&current_tokens_lookup_d,
             sizeof(TokenLookupElem)*fst_.numStates);
  bytes_cuda_malloc += sizeof(TokenLookupElem) * fst_.numStates;

  hipHostMalloc(&loglikelihoods_h, sizeof(BaseFloat) * (fst_.max_ilabel + 1));
  hipHostMalloc(&loglikelihoods_old_h, sizeof(BaseFloat) * (fst_.max_ilabel + 1));

  hipMalloc((void**)&loglikelihoods_d, sizeof(BaseFloat) * (fst_.max_ilabel + 1));
  bytes_cuda_malloc += sizeof(BaseFloat) * (fst_.max_ilabel + 1);
  hipMalloc((void**)&loglikelihoods_old_d,
             sizeof(BaseFloat) * (fst_.max_ilabel + 1));
  bytes_cuda_malloc += sizeof(BaseFloat) * (fst_.max_ilabel + 1);

  // for pruning
  bytes_cuda_malloc += lattice_processor_.Allocate(config.max_tokens_per_frame,
                       config.max_lat_arc_per_frame, config.prune_interval,
                       config.max_tokens, config.max_arcs, fst_);

  lat_arcs_buf_.Allocate(config.max_arcs, NULL, NULL, NULL,
                         lattice_processor_.GetDeviceArcsBpr());
  bytes_cuda_malloc += lat_arcs_buf_.GetCudaMallocBytes();

  for (int32 j = 0; j < LAT_BUF_SIZE; j++) {
    lat_toks_bufs_[j].Allocate(config.max_tokens_per_frame);
    bytes_cuda_malloc += lat_toks_bufs_[j].GetCudaMallocBytes();
  }

  // In each frame, we save the token
  // information in an array whose size is the number of arcs. This
  // ensures there are no write conflicts between threads since each
  // arc can be accessed at most once in each frame. It's a temp solution
  hipMalloc((void**)&token_per_arc_d, sizeof(Token)*config.max_lat_arc_per_frame);
  hipMalloc((void**)&token_per_arc_update_d,
             sizeof(int32)*config.max_lat_arc_per_frame); // temp solution
  hipMemset(token_per_arc_update_d, 0,
             sizeof(int32)*config.max_lat_arc_per_frame); // temp solution
  bytes_cuda_malloc += (sizeof(Token) + sizeof(int32)) *
                       (config.max_lat_arc_per_frame);

  num_frames_decoded_ = 0;
  UpdateTokPointersByFrame(num_frames_decoded_);

  hipStreamSynchronize(stream_comp);
  hipStreamSynchronize(stream_lat[0]);
  hipStreamSynchronize(hipStreamPerThread);
  // sgemm requires shared memory and we don't want cache config changing.
  // So set a device wide cache config.
  hipDeviceSetCacheConfig(hipFuncCachePreferEqual);
  if (config_.verbose > 1)
    get_free_memory_stat("After initlization:");
}

CudaLatticeDecoder::~CudaLatticeDecoder() {
  KALDI_VLOG(1) << "CUDA LatticeDecoder DESTRUCTOR\n";

  if (config_.verbose > 1)
    get_free_memory_stat("End of decoding:");

  for (int32 j = 0; j < LAT_BUF_SIZE; j++) lat_toks_bufs_[j].Free();
  lat_arcs_buf_.Free(true);
  lattice_processor_.Free();
  histogram_prev_toks_.Free();

  hipHostFree(loglikelihoods_h);
  hipHostFree(loglikelihoods_old_h);
  hipFree(loglikelihoods_d);
  hipFree(loglikelihoods_old_d);
  hipFree(current_tokens_lookup_d);

  hipFree(pe_idx_d);
  hipFree(agg_idx_d);
  hipFree(ne_idx_d);
  hipFree(ne_queue_d);
  hipFree(fb_idx_d);
  hipFree(barrier_d);

  hipFree(cutoff_d);
  hipFree(cutoff_prev_d);
  hipFree(num_arcs_till_last_d);
  hipFree(modified_d);

  hipFree(token_per_arc_d);
  hipFree(token_per_arc_update_d);

  hipEventDestroy(event_pt);
  hipEventDestroy(event_ll);
  hipStreamDestroy(stream_comp);
  for (int32 i = 0; i < LAT_BUF_SIZE; i++)
    hipStreamDestroy(stream_lat[i]);
  hipStreamDestroy(stream_ll);
}

void CudaLatticeDecoder::ComputeLogLikelihoods(DecodableInterface *decodable) {
  PUSH_RANGE("ComputeLogLikelihoods", 3)
  int32 frame = num_frames_decoded_;
  // finish decoding this frame, it has been ensured outside
  // hipStreamSynchronize(stream_comp);

  // double buffering so we don't overwrite loglikelihoods_h before it is copied down
  std::swap(loglikelihoods_h,
            loglikelihoods_old_h);
  std::swap(loglikelihoods_d, loglikelihoods_old_d);

  decodable->ComputeLogLikelihoods(loglikelihoods_h, frame, fst_.max_ilabel + 1);

  // copying in another stream to overlap transfer with compute
  hipMemcpyAsync(loglikelihoods_d, loglikelihoods_h,
                  sizeof(BaseFloat) * (fst_.max_ilabel + 1),
                  hipMemcpyHostToDevice, stream_ll);
  // mark log likelihoods are copied down to the device
  hipEventRecord(event_ll, stream_ll);

  // ensure logliklihoods_d is updated before consuming; we wait it in ProcessTokens
  // hipStreamWaitEvent(stream_comp,event_ll,0);
  POP_RANGE
}

// initialize parameters routine for launching cuda kernel
// GPU holds a local version of processTokens_params struct during launching
void CudaLatticeDecoder::InitParams(processTokens_params* params) {
  params->prev_toks = (*prev_toks_);
  params->cur_toks = (*cur_toks_);
  params->current_tokens_lookup = current_tokens_lookup_d;
  params->cutoff = cutoff_d;
  params->cutoff_prev = cutoff_prev_d;
  params->lat_arcs_sub_vec = lat_arcs_buf_;
  params->token_per_arc = token_per_arc_d;
  params->token_per_arc_update = token_per_arc_update_d;

  params->lattice_processor = lattice_processor_;
  params->histogram_prev_toks = histogram_prev_toks_;

  params->e_offsets = fst_.e_offsets_d;
  params->ne_offsets = fst_.ne_offsets_d;
  params->arc_ilabels = fst_.arc_ilabels_d;
  params->arc_olabels = fst_.arc_olabels_d;
  params->arc_weights = fst_.arc_weights_d;
  params->arc_nextstates = fst_.arc_nextstates_d;

  params->loglikelihoods = loglikelihoods_d;
  params->modified = modified_d;
  params->pe_idx = pe_idx_d;
  params->ne_idx = ne_idx_d;
  params->ne_queue = ne_queue_d;
  params->fb_idx = fb_idx_d;
  params->agg_idx = agg_idx_d;
  params->barrier = barrier_d;

  params->beam = config_.beam;
  params->verbose = config_.verbose;
  params->lattice_beam = config_.lattice_beam;
  params->prune_interval = config_.prune_interval;
  params->numArcs = fst_.NumArcs();
  params->frame = num_frames_decoded_;
  params->num_arcs_till_last = num_arcs_till_last_d;
  params->max_lat_arc_per_frame = config_.max_lat_arc_per_frame;
  params->max_active = config_.max_active;
}

// call InitDecoding if you have already decoded an
// utterance and want to start with a new utterance.
void CudaLatticeDecoder::InitDecoding() {
  if (config_.verbose > 1 ) KALDI_LOG << "CUDA LatticeDecoder InitDecoding\n";
  num_frames_decoded_ = 0;
  for (int32 i = 0; i < LAT_BUF_SIZE; i++) {
    ClearToks(lat_toks_bufs_[i]);
  }
  lat_arcs_buf_.Clear();

  UpdateTokPointersByFrame(num_frames_decoded_);
  lattice_processor_.Initialize();
  CU_SAFE_CALL(hipGetLastError());

  // we launch 64 threads as a block, i.e. 2 cooperative_groups
  // in cuda kernel of dynamic load balancing. more details are described there
  // we use a static launch size to reduce the kernel launch time 30us->10us
  int32 threads = 64;
  int32 blocks = DIV_ROUND_UP(total_threads, threads);

  // try to reduce number of tokens_allocation by not doing allocate, but only
  // init TokenLookupElem

  _initialize_all_states <<< blocks, threads, 0, stream_comp>>>(
    current_tokens_lookup_d, fst_.numStates, barrier_d);
  CU_SAFE_CALL(hipGetLastError());


  // initialize decoding:
  StateId start_state = fst_.Start();
  KALDI_ASSERT(start_state != fst::kNoStateId);

  processTokens_params params;
  InitParams(&params);
  _add_initial_token <<< 1, 1, 0, stream_comp>>>(params, start_state);
  CU_SAFE_CALL(hipGetLastError());

  ProcessNonemitting();

  if (config_.verbose > 1 ) KALDI_LOG <<
                                        "end of CUDA LatticeDecoder InitDecoding\n";
}

void CudaLatticeDecoder::UpdateTokPointersByFrame(uint32 frame) {
  cur_toks_ = &lat_toks_bufs_[frame % LAT_BUF_SIZE];
  prev_toks_ = &lat_toks_bufs_[(frame - 1) % LAT_BUF_SIZE];
  // single buffer in lat_arcs_buf_, so it doesn't need to do this
}

void CudaLatticeDecoder::ClearToks(TokenMergeVector &toks) {
  // cannot actually delete tokens as they are still used as lattice node
  toks.Clear(stream_comp);
}

void CudaLatticeDecoder::PreProcessTokens() {
  PUSH_RANGE("PreProcessTokens", 1)

  num_frames_decoded_++;
  UpdateTokPointersByFrame(num_frames_decoded_);
  ClearToks(*cur_toks_);
  // dont need to clear arcs as we directly take the final buffer into this vector

  POP_RANGE
}

void CudaLatticeDecoder::ProcessTokens() {
  PUSH_RANGE("ProcessTokens", 2)
  KALDI_VLOG(4) << num_frames_decoded_ << std::endl;

  // we launch 64 threads as a block, i.e. 2 cooperative_groups
  // in cuda kernel of dynamic load balancing. more details are described there
  // we use a static launch size to reduce the kernel launch time 30us->10us
  dim3 threads(64, 1);
  dim3 blocks(DIV_ROUND_UP(total_threads, (threads.x * threads.y)));
  if (num_frames_decoded_ == 1) KALDI_VLOG(2) << "# of blocks: " << blocks.x <<
        std::endl;

  // make sure log likelihoods are on the device before starting these kernels
  hipStreamWaitEvent(stream_comp, event_ll, 0);
  processTokens_params params;
  InitParams(&params);
  _process_tokens <<< blocks, threads, 0, stream_comp>>>(params); // doesn't work
  CU_SAFE_CALL(hipGetLastError());

  hipEventSynchronize(event_pt); // wait for last frame to finish
  hipEventRecord(event_pt, stream_comp);

  POP_RANGE
}

void CudaLatticeDecoder::ProcessNonemitting() {
  PUSH_RANGE("ProcessNonemitting", 0)

  // we launch 64 threads as a block, i.e. 2 cooperative_groups
  // in cuda kernel of dynamic load balancing. more details are described there
  // we use a static launch size to reduce the kernel launch time 30us->10us
  dim3 threads(64, 1);
  dim3 blocks(DIV_ROUND_UP(total_threads, (threads.x * threads.y)));

  processTokens_params params;
  InitParams(&params);
  _process_tokens <<< blocks, threads, 0, stream_comp>>>(params, true);
  CU_SAFE_CALL(hipGetLastError());

  POP_RANGE
}

// GPU lattice prune and copy the processed lattice nodes and arcs to host
void CudaLatticeDecoder::FinalProcessLattice(Token** toks_buf, int** toks_fr_sidx,
    LatLink** arcs_buf, int** arcs_fr_size, TokenMergeVector** toks_vec_last_fr) {
  PUSH_RANGE("FinalProcessLattice", 3)

  hipStreamSynchronize(stream_comp); // after fini comp. we can start copy
  // copy unpruned toks to host
  lattice_processor_.CopyToksToHost(num_frames_decoded_, stream_lat[0]);
  // GPU lattice pruning
  PruneActiveTokens(stream_comp, stream_comp, config_.lat_fraction);
  // copy the TokenState vector in the last frame, used by ComputeFinalCosts()
  CU_SAFE_CALL(hipGetLastError());
  (*cur_toks_).CopyDataToHost(stream_lat[1]);
  *toks_vec_last_fr = cur_toks_;
  hipStreamSynchronize(stream_comp); // wait for lattice pruning
  // copy pruned lattice arcs to host
  lattice_processor_.CopyArcsToHost(num_frames_decoded_, stream_lat[1]);
  // wait for all streams finishing
  hipStreamSynchronize(stream_lat[0]);
  hipStreamSynchronize(stream_lat[1]);
  // get host data from lattice_processor_, used by CPU lattice processing
  lattice_processor_.GetHostData(toks_buf, toks_fr_sidx,
                              arcs_buf, arcs_fr_size);
  CU_SAFE_CALL(hipGetLastError());

  KALDI_VLOG(1) << "Average tokens number, total frame: "
                << (*toks_fr_sidx)[num_frames_decoded_ + 1] / num_frames_decoded_
                << ", " << num_frames_decoded_;
  POP_RANGE
}

void CudaLatticeDecoder::PruneActiveTokens(hipStream_t wait_st,
    hipStream_t run_st, BaseFloat gpu_ratio) {
  // we launch 64 threads as a block, i.e. 2 cooperative_groups
  // in cuda kernel of dynamic load balancing. more details are described there
  // we use a static launch size to reduce the kernel launch time 30us->10us
  dim3 threads(64, 1);
  dim3 blocks(DIV_ROUND_UP(total_threads * gpu_ratio, (threads.x * threads.y)));
  hipStreamSynchronize(wait_st);
  if (config_.verbose > 1) KALDI_LOG << "PruneActiveTokens, # of blocks: " <<
                                       blocks.x << std::endl;
  processTokens_params params;
  InitParams(&params);
  _prune_active_tokens <<< blocks, threads, 0, run_st>>>(params);
}

// Outputs an FST corresponding to the single best path
// through the lattice. In lattice decoder, it is deprecated
bool CudaLatticeDecoder::GetBestPath(Lattice *fst_out,
                                     bool use_final_probs) const {
  KALDI_ERR << "We don't have this implementation in lattice decoder";
  return false;
}


} // end namespace kaldi.
